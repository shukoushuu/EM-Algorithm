#include "hip/hip_runtime.h"
#include <functional>

#include "auxiliares.cu"

using namespace std;

// Punteros a memoria global
double *g_datos;
double *g_resp;

double *g_verosimilitud;
double *g_verosimilitudParcial;
double *g_sumaProbabilidades;
double *g_medias;
double *g_pesos;
double *g_covarianzas;

double *g_L;
double *g_logDets;

__global__ void paso_e_cholesky(double *g_covarianzas, double *g_L, const size_t numDimensiones) // cholesky分解, 将对称正定的矩阵分解为一个下三角矩阵L及其转置的乘积. 这里分解的是协方差矩阵
{
    const size_t k = blockIdx.z;

    for (size_t j = 0; j < numDimensiones; j++) { // 先初始化为0
        for (size_t h = 0; h < numDimensiones; h++) {
            g_L[k * numDimensiones * numDimensiones + j * numDimensiones + h] = 0.0;
        }
    }

    for (size_t i = 0; i < numDimensiones; i++) {
        for (size_t j = 0; j < i + 1; j++) { // 只处理下三角的元素(包括对角线上的元素)
            double suma = 0.0;

            for (size_t h = 0; h < j; h++) {
                suma += g_L[k * numDimensiones * numDimensiones + i * numDimensiones + h] * g_L[k * numDimensiones * numDimensiones + j * numDimensiones + h];
            }

            g_L[k * numDimensiones * numDimensiones + i * numDimensiones + j] = (i == j) ?
                sqrt(g_covarianzas[k * numDimensiones * numDimensiones + i * numDimensiones + i] - suma) :
                (1.0 / g_L[k * numDimensiones * numDimensiones + j * numDimensiones + j] * (g_covarianzas[k * numDimensiones * numDimensiones + i * numDimensiones + j] - suma));
        }
    }
}

__global__ void paso_e(double *g_L, double *g_logDets, double *g_datos, double *g_pesos, double *g_medias, double *g_resp, const size_t n, size_t const numDimensiones) // 求log[N(x_i|mu_k, Sigma_k) * P(k)] (不包括d/2log(2*pi)那一项) 存储于g_resp
{
    const size_t i = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const size_t k = blockIdx.z;
    const size_t knd = k * numDimensiones;
    const size_t kndnd = knd * numDimensiones;

    if (i == 0 && threadIdx.x == 0) { // logarithmDeterminant 协方差矩阵Sigma行列式对数
       g_logDets[k] = logaritmoDeterminante(g_L, k, numDimensiones);
    }

    __syncthreads();

    if (i < n) { // 每个i对应一个thread, 对应一个样本
        extern __shared__ double sharedData[];
        double *v = (double*) &sharedData[threadIdx.x * numDimensiones]; // 将v的指针指向当前thread对应的数据
        double suma = 0.0;
        double tmp;

        for (size_t j = 0; j < numDimensiones; j++) { // 求v=L^(-1)(x_i - mu_k)
            tmp = g_datos[j * n + i] - g_medias[knd + j]; // 暂存(x_i - mu_k)[j]

            for (size_t h = 0; h < j; h++) {
                tmp -= g_L[kndnd + j * numDimensiones + h] * v[h];
            }

            v[j] = tmp / g_L[kndnd + j * numDimensiones + j];

            suma += v[j] * v[j];
        }

        g_resp[k * n + i] = -0.5 * (suma + g_logDets[k]) + log(g_pesos[k]); // log[N(x_i|mu_k, Sigma_k) * P(k)] (不包括d/2log(2*pi)那一项)
    }
}

__global__ void paso_e2(double *g_resp, double *g_verosimilitudParcial, const size_t n, const size_t numGaussianas) // 求p_ik存储于g_resp, 求log(Sigma(exp(z_k)))存储于g_verosimilitudParcial
{
    const size_t i = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (i < n) { // 每个i对应一个thread, 对应一个样本
        double suma = 0.0;
        double verosimilitudParcial;
        double maxContribucion = MENOS_INFINITO; // 对于给定的thread, 找到使p_ik最大的k, 先初始化为无穷小

        for (size_t k = 0; k < numGaussianas; k++) {
            if (g_resp[k * n + i] > maxContribucion) {
                maxContribucion = g_resp[k * n + i];
            }
        }

        for (size_t k = 0; k < numGaussianas; k++) {
            suma += exp(g_resp[k * n + i] - maxContribucion); // gpu中是column-major resp[k * n + i]也就是log[N(x_i|mu_k, Sigma_k) * P(k)] (不包括d/2log(2*pi)那一项), 对应下面的z_k
        }

        verosimilitudParcial = maxContribucion + log(suma); // 根据log-sum-exp公式 log(Sigma(exp(z_k))) = z_max + log(Sigma(exp(z_k - z_max))) verosimilitudParcial = log{Sigma[N(x_i|mu_k, Sigma_k) * P(k)]} (不包括d/2log(2*pi)那一项)

        for (size_t k = 0; k < numGaussianas; k++) {
            g_resp[k * n + i] = exp(g_resp[k * n + i] - verosimilitudParcial); // resp[k * n + i]重新赋值为[N(x_i|mu_k, Sigma_k) * P(k)] /Sigma[N(x_i|mu_k, Sigma_k) * P(k)], 这时才是真正的p_ik
        }

        g_verosimilitudParcial[i] = verosimilitudParcial; // log(Sigma(exp(z_k)))
    }
}

__global__ void paso_e_verosimilitud(double *g_verosimilitudParcial, double *g_verosimilitud, const size_t n) // n个样本的log(Sigma(exp(z_k)))规约获得最终的log(Likelihood)
{
    extern __shared__ double sharedData[];

    reducir<BLOCK_SIZE>([&] (size_t i) -> double { return g_verosimilitudParcial[i]; },
            [&] () -> double* { return &g_verosimilitud[blockIdx.x]; },
            [&] () -> void { reducirFinal<BLOCK_SIZE>([&] (size_t tid) -> double* { return &g_verosimilitud[tid]; }, [&] () -> double* { return &g_verosimilitud[0]; }, sharedData, gridDim.x); },
            n, sharedData, gridDim.x * gridDim.y * gridDim.z);
}

__global__ void paso_m(double *g_resp, double *g_sumaProbabilidades, double *g_pesos, const size_t n)
{
    extern __shared__ double sharedData[];

    const size_t k = blockIdx.z;

    const size_t numGaussianas = gridDim.z;

    reducir<BLOCK_SIZE>([&] (size_t i) -> double { return g_resp[k * n + i]; },
            [&] () -> double* { return &g_sumaProbabilidades[k * gridDim.x + blockIdx.x]; },
            [&] () -> void {
                for (size_t a = 0; a < numGaussianas; a++) {
                    reducirFinal<BLOCK_SIZE>([&] (size_t tid) -> double* { return &g_sumaProbabilidades[a * gridDim.x + tid]; }, [&] () -> double* { return &g_sumaProbabilidades[a]; }, sharedData, gridDim.x);
                    if (threadIdx.x == 0) g_pesos[a] = g_sumaProbabilidades[a] / n;
                }
            }, n, sharedData, gridDim.x * gridDim.z);
}

__global__ void paso_m2(double *g_resp, double *g_datos, double *g_sumaProbabilidades, double *g_medias, const size_t n)
{
    extern __shared__ double sharedData[];

    const size_t j = blockIdx.y;
    const size_t k = blockIdx.z;

    const size_t numGaussianas = gridDim.z;
    const size_t numDimensiones = gridDim.y;

    reducir<BLOCK_SIZE>([&] (size_t i) -> double { return g_resp[k * n + i] * g_datos[j * n + i]; },
            [&] () -> double* { return &g_medias[k * numDimensiones * gridDim.x + j * gridDim.x + blockIdx.x]; },
            [&] () -> void {
                for (size_t a = 0; a < numGaussianas; a++) {
                    for (size_t b = 0; b < numDimensiones; b++) {
                        reducirFinal<BLOCK_SIZE>([&] (size_t tid) -> double* { return &g_medias[a * numDimensiones * gridDim.x + b * gridDim.x + tid]; }, [&] () -> double* { return &g_medias[a * numDimensiones + b]; }, sharedData, gridDim.x);
                        if (threadIdx.x == 0) g_medias[a * numDimensiones + b] /= g_sumaProbabilidades[a];
                    }
                }
            }, n, sharedData, gridDim.x * gridDim.y * gridDim.z);
}

__global__ void paso_m_covarianzas(double *g_resp, double *g_datos, double *g_medias, double *g_covarianzas, const size_t n, const size_t numDimensiones)
{
    __shared__ double sharedData[BLOCK_SIZE];
    __shared__ size_t numBloques;
    __shared__ size_t j;
    __shared__ size_t h;
    __shared__ size_t k;
    __shared__ size_t kn;
    __shared__ size_t jn;
    __shared__ size_t hn;
    __shared__ size_t knd;
    __shared__ double medias_j;
    __shared__ double medias_h;

    if (threadIdx.x == 0) {
        numBloques = gridDim.x * gridDim.y * gridDim.z;
        j = blockIdx.y / numDimensiones;
        h = blockIdx.y % numDimensiones;
        k = blockIdx.z;
        kn = k * n;
        jn = j * n;
        hn = h * n;
        knd = k * numDimensiones;
        medias_j = g_medias[knd + j];
        medias_h = g_medias[knd + h];
    }

    __syncthreads();

    reducir<BLOCK_SIZE>([&] (size_t i) -> double { return g_resp[kn + i] * (g_datos[jn + i] - medias_j) * (g_datos[hn + i] - medias_h); },
            [&] () -> double* { return &g_covarianzas[knd * numDimensiones * gridDim.x + j * numDimensiones * gridDim.x + h * gridDim.x + blockIdx.x]; },
            [&] () -> void {
            }, n, sharedData, numBloques);
}

__global__ void paso_m_covarianzas_final(double *g_sumaProbabilidades, double *g_covarianzas, const size_t numTrozos)
{
    extern __shared__ double sharedData[];

    const size_t j = blockIdx.x;
    const size_t h = blockIdx.y;
    const size_t k = blockIdx.z;

    const size_t numDimensiones = gridDim.y;

    reducirFinal<BLOCK_SIZE>([&] (size_t tid) -> double* { return &g_covarianzas[k * numDimensiones * numDimensiones * numTrozos + j * numDimensiones * numTrozos + h * numTrozos + tid]; }, [&] () -> double* { return &g_covarianzas[k * numDimensiones * numDimensiones + j * numDimensiones + h]; }, sharedData, numTrozos);
    if (threadIdx.x == 0) g_covarianzas[k * numDimensiones * numDimensiones + j * numDimensiones + h] /= g_sumaProbabilidades[k];
}
